#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <memory>
#include <random>
#include <NvInfer.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace nvinfer1;

class Logger : public nvinfer1::ILogger {
public:
    void log(Severity severity, const char* msg) noexcept override {
        if (severity == Severity::kERROR || severity == Severity::kINTERNAL_ERROR) {
            std::cerr << "[ERROR] " << msg << std::endl;
        }
    }
};

// 读取 TensorRT engine 文件
vector<unsigned char> loadEngineFile(const string& enginePath) {
    ifstream file(enginePath, ios::binary);
    if (!file) {
        cerr << "Error: Could not open engine file: " << enginePath << endl;
        exit(EXIT_FAILURE);
    }
    file.seekg(0, ios::end);
    size_t size = file.tellg();
    file.seekg(0, ios::beg);
    vector<unsigned char> engineData(size);
    file.read(reinterpret_cast<char*>(engineData.data()), size);
    return engineData;
}

struct TRTDeleter {
    template <typename T>
    void operator()(T* obj) const {
        delete obj; 
    }
};

int main(int argc, char** argv) {
    std::string enginePath = "./rrdbnet_sim_quant_conv.trt";  // 直接指定 engine 文件路径

    // 加载 engine
    std::vector<unsigned char> engineData = loadEngineFile(enginePath);
    
    // 1. 读取 engine 文件
    Logger logger;

    unique_ptr<IRuntime, TRTDeleter> runtime(createInferRuntime(logger));
    unique_ptr<ICudaEngine, TRTDeleter> engine(runtime->deserializeCudaEngine(engineData.data(), engineData.size()));
    unique_ptr<IExecutionContext, TRTDeleter> context(engine->createExecutionContext());

    if (!context) {
        cerr << "Error: Failed to create execution context." << endl;
        return EXIT_FAILURE;
    }
    // auto runtime = unique_ptr<IRuntime>(createInferRuntime(logger));
    // auto engine = unique_ptr<ICudaEngine>(runtime->deserializeCudaEngine(engineData.data(), engineData.size()));
    // auto context = unique_ptr<IExecutionContext>(engine->createExecutionContext());

    // 2. 获取输入/输出 tensor 名称并分配 GPU 内存
    const char* inputTensorName = engine->getIOTensorName(0);
    const char* outputTensorName = engine->getIOTensorName(1);
    auto inputDims = context->getTensorShape(inputTensorName);
    auto outputDims = context->getTensorShape(outputTensorName);
    size_t tensorSize = 1 * 3 * 256 * 256 * sizeof(float); // 1x3x256x256

    vector<float> input_h(1 * 3 * 256 * 256);
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<float> dis(0.0f, 1.0f);
    for (size_t i = 0; i < input_h.size(); ++i) {
        input_h[i] = dis(gen);
    }
    vector<float> output_h(1 * 3 * 256 * 256);

    float* input_d;
    float* output_d;
    hipMalloc(&input_d, tensorSize);
    hipMalloc(&output_d, tensorSize);

    // 3. 创建 CUDA 流
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipMemcpyAsync(input_d, input_h.data(), tensorSize, hipMemcpyHostToDevice, stream);

    // 4. 运行推理
    float* bindings[] = {input_d, output_d};
    if (!context->executeV2(reinterpret_cast<void**>(bindings))) {
        cerr << "Error: Inference failed." << endl;
        return EXIT_FAILURE;
    }

    cout << "Inference completed successfully." << endl;

    hipMemcpyAsync(output_h.data(), output_d, tensorSize, hipMemcpyDeviceToHost, stream);
    
    // 同步流，确保所有操作完成
    hipStreamSynchronize(stream);

    // 5. 释放资源
    hipFree(input_d);
    hipFree(output_d);
    hipStreamDestroy(stream);
    return EXIT_SUCCESS;
}
